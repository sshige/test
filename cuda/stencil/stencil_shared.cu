// http://www.nvidia.com/docs/IO/116711/sc11-cuda-c-basics.pdf


#include <hip/hip_runtime.h>
#include <cstdio>
#include <iostream>

#define N 20
#define BLOCK_SIZE 10
#define RADIUS 2


__global__ void stencil_1d(int *in, int *out) {
  __shared__ int temp[BLOCK_SIZE + 2 * RADIUS];
  int gindex = threadIdx.x + blockIdx.x * blockDim.x;
  int lindex = threadIdx.x + RADIUS;

  // printf("blockIdx.x = %3d, threadIdx.x = %3d, blockDim.x=%3d\n", blockIdx.x, threadIdx.x, blockDim.x);
  printf("threadIdx.x = %3d, gindex = %3d, lindex = %3d\n",
         threadIdx.x, gindex, lindex);

  // Read Input elements into shared memory
  temp[lindex] = in[gindex];
  if (threadIdx.x < RADIUS) {
    temp[lindex - RADIUS] = in[gindex - RADIUS];
    temp[lindex + BLOCK_SIZE] = in[gindex + BLOCK_SIZE];
    printf("threadIdx.x = %3d, lindex - RADIUS = %3d, gindex - RADIUS = %3d, lindex + BLOCK_SIZE = %3d, gindex + BLOCK_SIZE = %3d,\nin[gindex - RADIUS] = %3d, in[gindex + BLOCK_SIZE] = %3d\n",
           threadIdx.x, lindex - RADIUS, gindex - RADIUS, lindex + BLOCK_SIZE, gindex + BLOCK_SIZE,
           in[gindex - RADIUS], in[gindex + BLOCK_SIZE]);
  }

  // Synchronize (ensure all the data is available)
  __syncthreads();

  // Apply the stencil
  int result = 0;
  for (int offset = -RADIUS; offset <= RADIUS; offset++) {
    result += temp[lindex + offset];
  }

  // Store the result
  out[gindex] = result;
}

void random_ints(int *a, int n) {
  for (int i = 0; i < n; i++) {
    a[i] = rand() % 10;
  }
}

void print_vector(int *a, int n) {
  printf("(");
  for (int i = 0; i < n; i++) {
    printf("%2d, ", a[i]);
  }
  printf(")\n");
}

int main(void) {
  int *a, *b; // host memory
  int *d_a, *d_b; // device memory
  int size = N * sizeof(int);

  // Allocate space for host memory of a, b
  a = (int *)malloc(size); random_ints(a, N);
  b = (int *)malloc(size);
  // Allocate space for device memory of a, b
  hipMalloc((void **)&d_a, size);
  hipMalloc((void **)&d_b, size);

  // Copy inputs to device
  hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);

  // Launch stencil_1d() kernel on GPU with N threads
  stencil_1d<<<N / BLOCK_SIZE, BLOCK_SIZE>>>(d_a, d_b);

  // Copy result back to host
  hipMemcpy(b, d_b, size, hipMemcpyDeviceToHost);

  std::cout << "a:" << std::endl;
  print_vector(a, N);
  std::cout << "b:" << std::endl;
  print_vector(b, N);

  // Cleanup
  free(a); free(b);
  hipFree(d_a); hipFree(d_b);

  return 0;
}
