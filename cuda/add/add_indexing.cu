// http://www.nvidia.com/docs/IO/116711/sc11-cuda-c-basics.pdf


#include <hip/hip_runtime.h>
#include <cstdio>
#include <ctime>
#include <cstdlib>
#include <iostream>


__global__ void add(int *a, int *b, int *c, int n) {
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  if (index < n) {
    c[index] = a[index] + b[index];
    // printf("index = %3d, blockIdx.x = %3d, threadIdx.x = %3d, blockDim.x = %3d\n",
    //        index, blockIdx.x, threadIdx.x, blockDim.x);
  }
}

void random_ints(int *a, int n) {
  for (int i = 0; i < n; i++) {
    a[i] = rand() % 20;
  }
}

void print_vector(int *a, int n) {
  printf("(");
  for (int i = 0; i < n; i++) {
    printf("%2d, ", a[i]);
  }
  printf(")\n");
}

int main(void) {
  int N = 2048 * 2048;
  int THREADS_PER_BLOCK = 512;

  int *a, *b, *c; // host memory
  int *d_a, *d_b, *d_c; // device memory
  int size = N * sizeof(int);

  // Allocate space for host memory of a, b, c
  srand(time(NULL));
  a = (int *)malloc(size); random_ints(a, N);
  b = (int *)malloc(size); random_ints(b, N);
  c = (int *)malloc(size);
  // Allocate space for device memory of a, b, c
  hipMalloc((void **)&d_a, size);
  hipMalloc((void **)&d_b, size);
  hipMalloc((void **)&d_c, size);

  // Copy inputs to device
  hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
  hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

  // Launch add() kernel on GPU with N threads
  add<<<(N + THREADS_PER_BLOCK - 1)/ THREADS_PER_BLOCK, THREADS_PER_BLOCK>>>(d_a, d_b, d_c, N);

  // Copy result back to host
  hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

  std::cout << "a:" << std::endl;
  print_vector(a, 10);
  std::cout << "b:" << std::endl;
  print_vector(b, 10);
  std::cout << "c:" << std::endl;
  print_vector(c, 10);

  // Cleanup
  free(a); free(b); free(c);
  hipFree(d_a); hipFree(d_b); hipFree(d_c);

  return 0;
}
