// http://www.nvidia.com/docs/IO/116711/sc11-cuda-c-basics.pdf


#include <hip/hip_runtime.h>
#include <cstdio>
#include <ctime>
#include <cstdlib>
#include <iostream>


__global__ void add(int *a, int *b, int *c) {
  c[threadIdx.x] = a[threadIdx.x] + b[threadIdx.x];
  printf("threadIdx.x = %3d\n", threadIdx.x);
}

void random_ints(int *a, int n) {
  for (int i = 0; i < n; i++) {
    a[i] = rand() % 20;
  }
}

void print_vector(int *a, int n) {
  printf("(");
  for (int i = 0; i < n; i++) {
    printf("%2d, ", a[i]);
  }
  printf(")\n");
}

int main(void) {
  int N = 10;
  // int N = 512;

  int *a, *b, *c; // host memory
  int *d_a, *d_b, *d_c; // device memory
  int size = N * sizeof(int);

  // Allocate space for host memory of a, b, c
  srand(time(NULL));
  a = (int *)malloc(size); random_ints(a, N);
  b = (int *)malloc(size); random_ints(b, N);
  c = (int *)malloc(size);
  // Allocate space for device memory of a, b, c
  hipMalloc((void **)&d_a, size);
  hipMalloc((void **)&d_b, size);
  hipMalloc((void **)&d_c, size);

  // Copy inputs to device
  hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
  hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

  // Launch add() kernel on GPU with N threads
  add<<<1,N>>>(d_a, d_b, d_c);

  // Copy result back to host
  hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

  std::cout << "a:" << std::endl;
  print_vector(a, N);
  std::cout << "b:" << std::endl;
  print_vector(b, N);
  std::cout << "c:" << std::endl;
  print_vector(c, N);

  // Cleanup
  free(a); free(b); free(c);
  hipFree(d_a); hipFree(d_b); hipFree(d_c);

  return 0;
}
